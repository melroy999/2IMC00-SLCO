#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include "example_system_gpuexplore.cuh"

#define MIN(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; })

#define MAX(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__inline__ void CUDA_CHECK_FOR_ERROR(int i) {
	//fprintf(stderr, "Call %u:\n", i);
	hipError_t err = hipGetLastError();
	CUDA_CHECK_RETURN(err);
}

int vmem = 0;

// Wrapper around hipMalloc to count allocated memory and check for error while allocating.
int cudaMallocCount ( void ** ptr,int size) {
	hipError_t err = hipSuccess;
	vmem += size;
	err = hipMalloc(ptr,size);
	if (err) {
		printf("Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);
		exit(1);
	}
	fprintf (stdout, "allocated %d\n", size);
	return size;
}

/**
 * CUDA kernel function to initialise the global memory hash table.
 */
__global__ void init_hash_table(compressed_nodetype *d_q, nodetype *d_q_i) {
    for (uint64_t i = GLOBAL_THREAD_ID; i < d_hash_table_size; i += NR_THREADS) {
    	d_q[i] = (compressed_nodetype) EMPTY_COMPRESSED_NODE;
    }
    for (uint64_t i = GLOBAL_THREAD_ID; i < d_internal_hash_table_size; i += NR_THREADS) {
    	d_q_i[i] = (nodetype) EMPTY_NODE;
    }
}

/**
 * CUDA kernel function to initialise the worktiles in the global memory.
 */
__global__ void init_worktiles(nodetype *d_worktiles) {
    if (THREAD_ID == 0) {
    	d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1)*BLOCK_ID + OPENTILELEN + LASTSEARCHLEN] = 0;
    }
    if (THREAD_ID < LASTSEARCHLEN) {
    	d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1)*BLOCK_ID + OPENTILELEN + THREAD_ID] = BLOCK_ID*BLOCK_SIZE + THREAD_ID*WARP_SIZE;
    }
    for (uint64_t i = THREAD_ID; i < OPENTILELEN; i += NR_THREADS) {
    	d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1)*BLOCK_ID + THREAD_ID] = EMPTYVECT32;
    }
}

/**
 * CUDA kernel function to count the number of states in the global memory hash table.
 */
__global__ void count_states(compressed_nodetype *d_q, uint64_t *result, uint64_t *new_result) {
 	__shared__ uint64_t blockcount, new_blockcount;

	if (THREAD_ID == 0) {
		blockcount = 0;
		new_blockcount = 0;
	}
	__syncthreads();
	uint64_t localResult = 0;
	uint64_t new_localResult = 0;
	for (uint64_t i = GLOBAL_THREAD_ID; i < d_hash_table_size; i += NR_THREADS) {
		if (d_q[i] != EMPTY_COMPRESSED_NODE) {
			localResult++;
			if (is_new(d_q[i])) {
				new_localResult++;
			}
		}
	}
	if (localResult > 0) {
		atomicAdd((unsigned long long *) &blockcount, (unsigned long long) localResult);
	}
	if (new_localResult > 0) {
		atomicAdd((unsigned long long *) &new_blockcount, (unsigned long long) new_localResult);
	}
	__syncthreads();
	if (THREAD_ID == 0) {
		if (blockcount > 0) {
			atomicAdd((unsigned long long *) result, (unsigned long long) blockcount);
		}
		if (new_blockcount > 0) {
			atomicAdd((unsigned long long *) new_result, (unsigned long long) new_blockcount);
		}
	}
}

/**
 * CUDA kernel function to count the number of internal nodes in the global memory internal hash table.
 */
__global__ void count_internal_nodes(nodetype *d_q_i, uint64_t *result) {
 	__shared__ uint64_t blockcount;

	if (THREAD_ID == 0) {
		blockcount = 0;
	}
	__syncthreads();
	uint64_t localResult = 0;
	for (uint64_t i = GLOBAL_THREAD_ID; i < d_internal_hash_table_size; i += NR_THREADS) {
		if (d_q_i[i] != EMPTY_NODE) {
			localResult++;
		}
	}
	if (localResult > 0) {
		atomicAdd((unsigned long long *) &blockcount, (unsigned long long) localResult);
	}
	__syncthreads();
	if (THREAD_ID == 0) {
		if (blockcount > 0) {
			atomicAdd((unsigned long long *) result, (unsigned long long) blockcount);
		}
	}
}

/**
 * CUDA kernel function to prepare the cache for a new successor generation iteration.
 */
inline __device__ void PREPARE_CACHE() {
	// Traverse the new state vector trees, and reconstruct the nodes, i.e., put the cache pointers back in place and reconstruct the non-leaf nodes
	// by inspecting their children.
	shared_inttype pointers;
	shared_indextype addr;
	nodetype node;
	bool next_it, is_required;

	// First we mark the root nodes referred to in the worktile for preparation.
	for (shared_indextype i = THREAD_ID; i < OPENTILECOUNT; i += BLOCK_SIZE) {
		mark_cached_node_as_next_in_preparation(&shared[CACHEOFFSET+(shared[OPENTILEOFFSET+(2*i)+1]*3)+2]);
	}

	while (CONTINUE == 1) {
		__syncthreads();
		if (THREAD_ID == 0) {
			CONTINUE = 0;
		}
		__syncthreads();
		#pragma unroll
		for (shared_indextype i = THREAD_ID; (i*3)+2 < d_shared_size - CACHEOFFSET; i += BLOCK_SIZE) {
			if (cached_node_is_next_in_preparation(shared[CACHEOFFSET+(i*3)+2])) {
				node = combine_halfs(shared[CACHEOFFSET+(i*3)], shared[CACHEOFFSET+(i*3)+1]);
				is_required = false;
				if (!is_root(node)) {
					is_required = cached_node_is_required(shared[CACHEOFFSET+(i*3)]);
					// Put the original cache pointers back if the node is new, i.e., it is not yet set as required.
					if (!is_required) {
						shared[CACHEOFFSET+(i*3)+2] = shared[CACHEOFFSET+(i*3)];
					}
				}
				// Mark its children for reconstruction and reconstruct the node, if needed.
				next_it = false;
				addr = sv_step(i, false);
				pointers = shared[CACHEOFFSET+(addr*3)+2];
				if (!cached_node_is_leaf_with_global_address(pointers)) {
					// By definition, a left child stores a global address in its cache pointers.
					// Is the node not a global address stub?
					if (shared[CACHEOFFSET+(addr*3)] != EMPTYVECT32) {
						mark_cached_node_as_next_in_preparation(&shared[CACHEOFFSET+(addr*3)+2]);
						next_it = true;
					}
				}
				if (!is_required) {
					set_left_in_vectortree_node(&node, global_address(pointers));
					// Reset left cache pointer in case the left child is a stub.
					if (shared[CACHEOFFSET+(addr*3)] == EMPTYVECT32) {
						set_left_cache_pointer((shared_inttype *) &shared[CACHEOFFSET+(i*3)+2], EMPTY_CACHE_POINTER);
					}
				}
				addr = sv_step(i, true);
				// Is there actually a right child?
				if (addr != EMPTY_CACHE_POINTER) {
					pointers = shared[CACHEOFFSET+(addr*3)+2];
					if (!cached_node_is_leaf_with_global_address(pointers)) {
						if (!cached_node_contains_global_address(pointers)) {
							mark_cached_node_as_required(&shared[CACHEOFFSET+(addr*3)]);
						}
						mark_cached_node_as_next_in_preparation(&shared[CACHEOFFSET+(addr*3)+2]);
						next_it = true;
					}
				}
				if (!is_required) {
					// Store the node.
					shared[CACHEOFFSET+(i*3)] = get_left(node);
					shared[CACHEOFFSET+(i*3)+1] = get_right(node);
				}
				mark_cached_node_as_required(&shared[CACHEOFFSET+(i*3)]);
				mark_cached_node_as_old(&shared[CACHEOFFSET+(i*3)+2]);
				if (next_it) {
					// A next iteration is required.
					CONTINUE = 1;
				}
			}
		}
		__syncthreads();
	}
	// Scan the cache one more time, remove non-leaf nodes that are no longer required (alternative: keep them with their global memory addresses)
	// and reset the 'required' marks of required non-leaf nodes.
	#pragma unroll
	for (shared_indextype i = THREAD_ID; (i*3)+2 < d_shared_size - CACHEOFFSET; i += BLOCK_SIZE) {
		pointers = shared[CACHEOFFSET+(i*3)+2];
		if (pointers != EMPTYVECT32) {
			if (!cached_node_is_leaf_with_global_address(pointers)) {
				if (cached_node_is_required(shared[CACHEOFFSET+(i*3)])) {
					reset_cached_node_required(&shared[CACHEOFFSET+(i*3)]);
				}
				else {
					// Delete node.
					shared[CACHEOFFSET+(i*3)] = EMPTYVECT32;
					shared[CACHEOFFSET+(i*3)+1] = EMPTYVECT32;
					shared[CACHEOFFSET+(i*3)+2] = EMPTYVECT32;
				}
			}
		}
	}
}

__global__ void __launch_bounds__(512, 2) gather(compressed_nodetype *d_q, nodetype *d_q_i, bool *d_dummy, uint8_t *d_contBFS, uint8_t *d_property_violation, volatile uint8_t *d_newstate_flags, nodetype *d_worktiles, const uint8_t scan) {
	uint64_t i;
	indextype l;
	shared_indextype sh_index, opentile_scan_start;
	nodetype tmp;

	// Reset the shared variables preceding the cache and reset the cache.
	if (THREAD_ID < SH_OFFSET) {
		shared[THREAD_ID] = 0;
	}
	for (i = THREAD_ID; i < (d_shared_size - SH_OFFSET); i += BLOCK_SIZE) {
		shared[SH_OFFSET+i] = EMPTYVECT32;
	}
	if (scan) {
		__syncthreads();
		// Copy the work tile from global memory.
		if (THREAD_ID < OPENTILELEN + LASTSEARCHLEN + 1) {
			i = d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1) * BLOCK_ID + THREAD_ID];
			if (THREAD_ID < OPENTILELEN) {
				shared[OPENTILEOFFSET + (2*THREAD_ID)] = get_left(i);
				shared[OPENTILEOFFSET + (2*THREAD_ID) + 1] = get_right(i);
			}
			else if (THREAD_ID < OPENTILELEN + LASTSEARCHLEN) {
				shared[LASTSEARCHOFFSET + THREAD_ID - OPENTILELEN] = (shared_inttype) i;				
			}
			else {
				OPENTILECOUNT = (shared_inttype) i;
			}
		}
	}
	__syncthreads();
	while (ITERATIONS < d_kernel_iters) {
		if (ITERATIONS > 0) {
			// Prepare the cache for the next iteration.
			PREPARE_CACHE();
			__syncthreads();
		}
		if (THREAD_ID == 0 && OPENTILECOUNT < OPENTILELEN && d_newstate_flags[BLOCK_ID] == 1) {
			// Indicate that we are scanning.
			d_newstate_flags[BLOCK_ID] = 0;
			SCAN = 1;
		}
		// We store the current value of OPENTILECOUNT in opentile_scan_start, to check later whether we have added scanned states
		// to a non-empty work-tile, and to identify those newly added states for fetching. If this is the first iteration, this is
		// not relevant, as in that case, all states are newly added and require fetching.
		opentile_scan_start = 0;
		if (ITERATIONS > 0) {
			opentile_scan_start = OPENTILECOUNT;
		}
		__syncthreads();
		// Scan the open set for work; we use OPENTILECOUNT to count retrieved elements.
		if (SCAN) {
			uint64_t loc = (uint64_t) shared[LASTSEARCHOFFSET + WARP_ID] + LANE;
			// This block should be able to find a new state.
			for (i = (GLOBAL_WARP_ID*WARP_SIZE); i < d_hash_table_size && OPENTILECOUNT < OPENTILELEN; i += NR_THREADS) {
				if (loc < d_hash_table_size) {
					tmp = d_q[loc];
					if (is_new(tmp)) {
						// Try to increment the OPENTILECOUNT counter. If successful, store a reference to the state.
						l = atomicAdd((shared_inttype *) &OPENTILECOUNT, 1);
						if (l < OPENTILELEN) {
							d_q[loc] = mark_old(tmp);
							tmp = get_uncompressed_node_root(tmp, loc);
							shared[OPENTILEOFFSET+(2*l)] = get_left(tmp);
							shared[OPENTILEOFFSET+(2*l)+1] = get_right(tmp);
						}
					}
				}
				loc += NR_THREADS;
				if (loc >= d_hash_table_size) {
					loc = GLOBAL_THREAD_ID;
				}
			}
			if (LANE == 0) {
				shared[LASTSEARCHOFFSET + WARP_ID] = loc;
			}
		}
		__syncthreads();
		// If work has been retrieved, indicate this.
		if (THREAD_ID == 0) {
			if (OPENTILECOUNT > 0) {
				(*d_contBFS) = 1;
				if (OPENTILECOUNT > OPENTILELEN) {
					OPENTILECOUNT = OPENTILELEN;
				}
			}
			if (SCAN && OPENTILECOUNT == OPENTILELEN) {
				// Scanning has completed and the open tile has been filled with new states.
				// There may still be more new states to be retrieved.
                d_newstate_flags[BLOCK_ID] = 1;
			}
		}
		if (OPENTILECOUNT > opentile_scan_start) {
			// Fill the cache with the newly added vector trees referred to in the work tile.
			// Create a cooperative group within a warp in which the thread resides.
			thread_block_tile<VECTOR_GROUP_SIZE> gtile = tiled_partition<VECTOR_GROUP_SIZE>(this_thread_block());

			#pragma unroll
			for (i = VECTOR_GROUP_ID; i < (OPENTILECOUNT-opentile_scan_start); i += NR_VECTOR_GROUPS_PER_BLOCK) {
				l = FETCH(gtile, d_q, d_q_i, opentile_scan_start+i);
				if (l == CACHE_FULL) {
					// PLAN B?
				}
				else {
					sh_index = (shared_indextype) l;
				}
				if (gtile.thread_rank() == 0) {
					// Store the address to the tree in the cache in the work tile.
					shared[OPENTILEOFFSET+2*(opentile_scan_start+i)] = 0;
					shared[OPENTILEOFFSET+2*(opentile_scan_start+i)+1] = sh_index;
				}
			}
		}
		__syncthreads();
		if (GENERATE_SUCCESSORS(d_q, d_q_i, d_dummy, d_newstate_flags) == HASH_TABLE_FULL) {
			CONTINUE = 2;
		}
		bool performed_work = OPENTILECOUNT != 0;
		__syncthreads();
		// Reset the work tile count
		if (THREAD_ID == 0) {
			OPENTILECOUNT = 0;
		}
		// Start scanning the local cache and write results to the global hash table.
		if (performed_work) {
			FINDORPUT_MANY(d_q, d_q_i, d_dummy, d_newstate_flags);
		}
		__syncthreads();
		// Write 'empty' pointers to unused part of the work tile.
		if (THREAD_ID < 2*(OPENTILELEN - OPENTILECOUNT)) {
			shared[OPENTILEOFFSET+2*(OPENTILECOUNT)+THREAD_ID] = EMPTYVECT32;
		}
		// Ready to start next iteration, if error has not occurred.
		if (THREAD_ID == 0) {
			if (CONTINUE == 2) {
				(*d_contBFS) = 2;
				ITERATIONS = d_kernel_iters;
			}
			else if (CONTINUE == 3) {
				(*d_contBFS) = 3;
				ITERATIONS = d_kernel_iters;
			}
			else {
				ITERATIONS++;
			}
			CONTINUE = 1;
		}
		__syncthreads();
	}
	// Done. Copy the work tile to global memory.
	if (THREAD_ID < OPENTILELEN+LASTSEARCHLEN+1) {
		if (THREAD_ID < OPENTILELEN) {
			i = combine_halfs(shared[OPENTILEOFFSET+(2*THREAD_ID)], shared[OPENTILEOFFSET+(2*THREAD_ID)+1]);
		}
		else if (THREAD_ID < OPENTILELEN + LASTSEARCHLEN) {
			i = shared[LASTSEARCHOFFSET+THREAD_ID-OPENTILELEN];			
		}
		else {
			i = OPENTILECOUNT;
		}
		d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1)*BLOCK_ID + THREAD_ID] = i;
	}
}

/**
 * Host function that prepares data, copies it to the GPU, and handles the control flow of the model checking.
 */
int main(int argc, char** argv) {
	// Size of global hash table.
	uint64_t hash_table_size;
	// Size of the internal hash table.
	uint64_t internal_hash_table_size;
	// Number of search iterations in each kernel launch.
	uint32_t kernel_iters = KERNEL_ITERS;
	// Level of verbosity (1=print level progress)
	int verbosity = 0;
	// Clock to measure time.
	clock_t start, stop;
	double runtime = 0.0;

	// Start timer.
	assert((start = clock()) != -1);

	hipDeviceProp_t prop;
	int nDevices;

	// Flag to keep track of the progress and whether hash table errors occurred (value == 2).
	uint8_t contBFS, *d_contBFS;
	// Flags to track which blocks have new states.
	uint8_t *d_newstate_flags;
	// Flag to keep track of property verification outcome.
	uint8_t *d_property_violation;
	// Integer to store the amount of states counted in the hash table.
	uint64_t counted_states, *d_counted_states;
	// Integer to store the amount of new states counted in the hash table.
	uint64_t new_counted_states, *d_new_counted_states;
	// Space to temporarily store work tiles.
	nodetype *d_worktiles;

	// Global hash table.
	compressed_nodetype *d_q;
	// Internal node global hash table.
	nodetype *d_q_i;
	// Dummy flag to regulate writes to d_q_i.
	bool *d_dummy;

	const char* help_text =
		"Usage: gpuexplore [OPTIONS]\n"
		"Run state-space exploration on preprocessed SLCO model.\n"
		"options:\n"
		"  -k NUM           Run NUM iterations per kernel launch (default 1).\n"
		"  -q NUM           Allocate NUM integers for the global hash table (default fill the memory).\n"
		"  -v NUM           Change the verbosity level:\n"
		"                      0 - minimal output.\n"
		"                      1 - print sequence number of each kernel launch (search step).\n"
		"                      2 - print number of states in the global hash table after each kernel launch.\n"
		"                      3 - print global hash table content after each kernel launch.\n"
		"                      4 - print global hash table content, with pointer info, after each kernel launch.\n"
		"  -h, --help, -?   Show this help message.\n";

	int i = 1;
	while (i < argc) {
		if (!strcmp(argv[i],"--help") || !strcmp(argv[i],"-h") || !strcmp(argv[i],"-?")) {
			fprintf(stdout, "%s", help_text);
			exit(0);
		}
		else if (!strcmp(argv[i],"-k")) {
			kernel_iters = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-q")) {
			hash_table_size = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-v")) {
			verbosity = atoi(argv[i+1]);
			if (verbosity > 6) {
				verbosity = 6;
			}
			i += 2;
		}
		else {
			fprintf(stderr, "ERROR: unrecognised option %s!\n", argv[i]);
			fprintf(stdout, "%s", help_text);
			exit(1);
		}
	}

	// Set continue flag.
	contBFS = 1;
	
	// Query the device properties and determine the data structure sizes.
	hipGetDeviceCount(&nDevices);
	if (nDevices == 0) {
		fprintf(stderr, "ERROR: No CUDA compatible GPU detected!\n");
		exit(1);
	}
	hipGetDeviceProperties(&prop, 0);
	fprintf (stdout, "global mem: %lu\n", (uint64_t) prop.totalGlobalMem);
	fprintf (stdout, "shared mem per block: %d\n", (int) prop.sharedMemPerBlock);
	fprintf (stdout, "shared mem per SM: %d\n", (int) prop.sharedMemPerMultiprocessor);
	fprintf (stdout, "max. threads per block: %d\n", (int) prop.maxThreadsPerBlock);
	fprintf (stdout, "max. grid size: %d\n", (int) prop.maxGridSize[0]);
	fprintf (stdout, "nr. of multiprocessors: %d\n", (int) prop.multiProcessorCount);

	// Determine actual number of blocks.
	uint32_t nblocks = MAX(1,MIN(prop.maxGridSize[0], NR_BLOCKS));

	// Allocate memory on the GPU.
	cudaMallocCount((void **) &d_contBFS, sizeof(uint8_t));
	cudaMallocCount((void **) &d_property_violation, sizeof(uint8_t));
	cudaMallocCount((void **) &d_counted_states, sizeof(uint64_t));
	cudaMallocCount((void **) &d_new_counted_states, sizeof(uint64_t));
	cudaMallocCount((void **) &d_newstate_flags, nblocks * sizeof(uint8_t));
	cudaMallocCount((void **) &d_worktiles, nblocks * (OPENTILELEN+LASTSEARCHLEN+1) * sizeof(nodetype));
	cudaMallocCount((void **) &d_dummy, sizeof(bool));

	// Set data on the GPU to initial values.
	CUDA_CHECK_RETURN(hipMemset(d_contBFS, 1, sizeof(uint8_t)));
	CUDA_CHECK_RETURN(hipMemset(d_newstate_flags, 0, nblocks * sizeof(uint8_t)));
	CUDA_CHECK_RETURN(hipMemset(d_counted_states, 0, sizeof(uint64_t)));
	CUDA_CHECK_RETURN(hipMemset(d_new_counted_states, 0, sizeof(uint64_t)));

	// We create a global compact hash table for 24 GB. A root table is created that has exactly 2^32 elements, and an internal table is created with 500 million elements.
	hash_table_size = 4294967296;
	hipMalloc((void **)&d_q, hash_table_size * sizeof(compressed_nodetype));
	internal_hash_table_size = 536870912;
	hipMalloc((void **)&d_q_i, internal_hash_table_size * sizeof(nodetype));

	fprintf (stdout, "Global mem hash table size: %lu; Number of entries: %lu\n", hash_table_size*sizeof(compressed_nodetype),  hash_table_size);
	fprintf (stdout, "Internal global mem hash table size: %lu; Number of entries: %lu\n", internal_hash_table_size*sizeof(nodetype), internal_hash_table_size);

	// The size of the shared caches is set to a precomputed value.
	shared_inttype shared_size = 11925+CACHEOFFSET;
	fprintf (stdout, "Shared mem work tile size: 170\n");
	fprintf (stdout, "Shared mem size: %u; Number of entries in the cache: %u\n", (uint32_t) (shared_size*sizeof(shared_inttype)), (uint32_t) (shared_size - CACHEOFFSET)/3);
	fprintf (stdout, "Nr. of blocks: %d; Block size: 512; Nr. of kernel iterations: %d\n", nblocks, kernel_iters);

	// Copy symbols.
	hipMemcpyToSymbol(HIP_SYMBOL(d_shared_size), &shared_size, sizeof(shared_inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_iters), &kernel_iters, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_internal_hash_table_size), &internal_hash_table_size, sizeof(uint64_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_hash_table_size), &hash_table_size, sizeof(uint64_t));

	// Initialise the hash table.
	init_hash_table<<<nblocks, 512>>>(d_q, d_q_i);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	// Initialise the worktiles.
	init_worktiles<<<nblocks, 512>>>(d_worktiles);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	// Store the initial state.
	store_initial_state<<<nblocks, 512, shared_size * sizeof(shared_inttype)>>>(d_q, d_q_i, d_dummy, d_newstate_flags, d_worktiles);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	compressed_nodetype *q_test;
	nodetype *q_i_test;
	if (verbosity >= 3) {
		q_test = (compressed_nodetype*) malloc(sizeof(compressed_nodetype)*hash_table_size);
		q_i_test = (nodetype*) malloc(sizeof(nodetype)*internal_hash_table_size);
	}
	// Create a vector to store the states in a sorted way (for verbosity levels >= 5).
	std::vector<systemstate_t> states;

	uint32_t iterations_counter = 0;
	uint8_t scan = 1;
	CUDA_CHECK_RETURN(hipMemset(d_property_violation, 0, sizeof(uint8_t)));
	uint8_t property_violation = 0;

	clock_t exploration_start;
	assert((exploration_start = clock()) != -1);

	while (contBFS == 1) {
		CUDA_CHECK_RETURN(hipMemset(d_contBFS, 0, sizeof(uint8_t)));
		// To investigate: changing the size of the shared mem: hipFuncSetAttribute(reinterpret_cast<const void*>(gather), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
		gather<<<nblocks, 512, shared_size * sizeof(shared_inttype)>>>(d_q, d_q_i, d_dummy, d_contBFS, d_property_violation, d_newstate_flags, d_worktiles, scan);

		// Copy progress result.
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&contBFS, d_contBFS, sizeof(uint8_t), hipMemcpyDeviceToHost));
		// if (check_property > 0) {
		// }
		if (verbosity > 0) {
			if (verbosity == 1) {
				fprintf(stdout, "%d\n", iterations_counter++);
			}
			else if (verbosity == 2) {
				hipMemset(d_counted_states, 0, sizeof(uint64_t));
				hipMemset(d_new_counted_states, 0, sizeof(uint64_t));
				count_states<<<((int) prop.multiProcessorCount)*8, 512>>>(d_q, d_counted_states, d_new_counted_states);
				CUDA_CHECK_RETURN(hipDeviceSynchronize());
				CUDA_CHECK_RETURN(hipMemcpy(&counted_states, d_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
				CUDA_CHECK_RETURN(hipMemcpy(&new_counted_states, d_new_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
				fprintf(stdout, "Nr. of states in hash table: %lu, new states: %lu\n", counted_states, new_counted_states);
			}
			else if (verbosity == 3) {
				hipMemcpy(q_test, d_q, hash_table_size * sizeof(compressed_nodetype), hipMemcpyDeviceToHost);
				hipMemcpy(q_i_test, d_q_i, internal_hash_table_size * sizeof(nodetype), hipMemcpyDeviceToHost);
				print_content_hash_table(stdout, q_test, q_i_test, hash_table_size, internal_hash_table_size, false);
			}
			else if (verbosity == 4) {
				hipMemcpy(q_test, d_q, hash_table_size * sizeof(compressed_nodetype), hipMemcpyDeviceToHost);
				hipMemcpy(q_i_test, d_q_i, internal_hash_table_size * sizeof(nodetype), hipMemcpyDeviceToHost);
				print_content_hash_table(stdout, q_test, q_i_test, hash_table_size, internal_hash_table_size, true);
			}
			else if (verbosity == 5) {
				// Produce a sorted list of states at the end of the state space generation.
				// First count the number of states.
				CUDA_CHECK_RETURN(hipMemset(d_counted_states, 0, sizeof(uint64_t)));
				CUDA_CHECK_RETURN(hipMemset(d_new_counted_states, 0, sizeof(uint64_t)));
				count_states<<<((int) prop.multiProcessorCount)*8, 512>>>(d_q, d_counted_states, d_new_counted_states);
				CUDA_CHECK_RETURN(hipDeviceSynchronize());
				CUDA_CHECK_RETURN(hipMemcpy(&counted_states, d_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
				CUDA_CHECK_RETURN(hipMemcpy(&new_counted_states, d_new_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
				fprintf(stdout, "Nr. of states in hash table: %lu, new states: %lu\n", counted_states, new_counted_states);
				states.clear();
				states.reserve(counted_states);
				// Scan the hash table for states.
				hipMemcpy(q_test, d_q, hash_table_size * sizeof(compressed_nodetype), hipMemcpyDeviceToHost);
				for (indextype i = 0; i < hash_table_size; i++) {
					if (is_new(q_test[i])) {
						states.push_back(get_systemstate(q_test, i, q_i_test));
					}
				}
				// Sort the states.
				std::sort(states.begin(), states.end(), systemstates_compare);
				// Print the states compactly.
				print_systemstates(stdout, states);
			}
		}
		scan = 1;
	}

	// Determine runtime.
	stop = clock();
	runtime = (double) (stop-start)/CLOCKS_PER_SEC;
	fprintf(stdout, "Run time: %f\n", runtime);
	runtime = (double) (stop-exploration_start)/CLOCKS_PER_SEC;
	fprintf(stdout, "Exploration time: %f\n", runtime);

	// TODO: Property violation report

	// Report hash table error if required.
	if (contBFS == 2) {
		fprintf(stderr, "ERROR: root hash table considered full!\n");
	}
	else if (contBFS == 3) {
		fprintf(stderr, "ERROR: internal hash table considered full!\n");
	}

	if (verbosity == 6) {
		// Produce a sorted list of states at the end of the state space generation.
		// First count the number of states.
		CUDA_CHECK_RETURN(hipMemset(d_counted_states, 0, sizeof(uint64_t)));
		CUDA_CHECK_RETURN(hipMemset(d_new_counted_states, 0, sizeof(uint64_t)));
		count_states<<<((int) prop.multiProcessorCount)*8, 512>>>(d_q, d_counted_states, d_new_counted_states);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&counted_states, d_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&new_counted_states, d_new_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
		fprintf(stdout, "Nr. of states in hash table: %lu, new states: %lu\n", counted_states, new_counted_states);
		// Create a vector to store the states in a sorted way.
		std::vector<systemstate_t> states;
		states.reserve(counted_states);
		// Scan the hash table for states.
		hipMemcpy(q_test, d_q, hash_table_size * sizeof(compressed_nodetype), hipMemcpyDeviceToHost);
		hipMemcpy(q_i_test, d_q_i, internal_hash_table_size * sizeof(nodetype), hipMemcpyDeviceToHost);
		for (uint64_t i = 0; i < hash_table_size; i++) {
			if (q_test[i] != EMPTY_COMPRESSED_NODE) {
				states.push_back(get_systemstate(q_test, i, q_i_test));
			}
		}
		// Sort the states.
		std::sort(states.begin(), states.end(), systemstates_compare);
		// Print the states compactly.
		print_systemstates(stdout, states);
	}
	else {
		CUDA_CHECK_RETURN(hipMemset(d_counted_states, 0, sizeof(uint64_t)));
		CUDA_CHECK_RETURN(hipMemset(d_new_counted_states, 0, sizeof(uint64_t)));
		count_states<<<((int) prop.multiProcessorCount)*8, 512>>>(d_q, d_counted_states, d_new_counted_states);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&counted_states, d_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&new_counted_states, d_new_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
		fprintf(stdout, "Nr. of states in hash table: %lu, new states: %lu\n", counted_states, new_counted_states);

		CUDA_CHECK_RETURN(hipMemset(d_counted_states, 0, sizeof(uint64_t)));
		count_internal_nodes<<<((int) prop.multiProcessorCount)*8, 512>>>(d_q_i, d_counted_states);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&counted_states, d_counted_states, sizeof(uint64_t), hipMemcpyDeviceToHost));
		fprintf(stdout, "Nr. of internal nodes: %lu\n", counted_states);
	}

	return 0;
}
